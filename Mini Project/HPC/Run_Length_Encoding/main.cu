#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <unistd.h>
#include <vector>
#include <cstring>
#include <string>
#include <time.h>
#include "hemi/hemi.h"
#include "hemi/kernel.h"
#include "hemi/parallel_for.h"
#include "hemi/launch.h"
#include "cub/util_allocator.cuh"
#include "cub/device/device_scan.cuh" 
#include "cub/device/device_run_length_encode.cuh"
using in_elt_t = int;

float parallel_elapsed_time = 0.0;
hipEvent_t gpu_start, gpu_stop;

template<typename elt_t>
struct array
{
	elt_t *data;
	size_t size; // the number of elt_t elements in data

	static array<elt_t> new_on_device(size_t size)
	{
		array<elt_t> d_result{nullptr, size};
		d_result.hipMalloc();
		return d_result;
	}

	static array<elt_t> vector_view_on_host(std::vector<elt_t> &v)
	{
		return array<elt_t>{v.data(), v.size()};
	}

	array<elt_t> subview(size_t offset, size_t subview_size)
	{
		size_t result_size = std::min(subview_size, size - offset);
		return array<elt_t>{data + offset, result_size};
	}

	elt_t &operator[](const size_t i)
	{
		return data[i];
	}

	void hipMalloc()
	{
		checkCuda(::hipMalloc(&data, size * sizeof(*data)));
	}

	void hipFree()
	{
		checkCuda(::hipFree(data));
	}
};

void append_partial_result(std::vector<in_elt_t> &out_symbols, std::vector<int> &out_counts, std::vector<in_elt_t> &full_out_symbols, std::vector<int> &full_out_counts) {
	size_t offset = 0;

	if (full_out_symbols.size() > 0 && out_symbols.size() > 0) {
		size_t prev_full_end{full_out_symbols.size() - 1};
		if (full_out_symbols[prev_full_end] == out_symbols[0]) {
			full_out_counts[prev_full_end] += out_counts[0];
			offset = 1;
		}
	}

	std::copy(out_symbols.begin() + offset, out_symbols.end(), std::back_inserter(full_out_symbols));
	std::copy(out_counts.begin() + offset, out_counts.end(), std::back_inserter(full_out_counts));
}

int serial_rle_helper(const in_elt_t* in, int n, in_elt_t* symbolsOut, int* countsOut) {
	if (n == 0)
		return 0; // nothing to compress!

	int outIndex = 0;
	in_elt_t symbol = in[0];
	int count = 1;

	for (int i = 1; i < n; ++i) {
		if (in[i] != symbol) {
			// run is over.
			// So output run.
			symbolsOut[outIndex] = symbol;
			countsOut[outIndex] = count;
			outIndex++;

			// and start new run:
			symbol = in[i];
			count = 1;
		} else {
			++count; // run is not over yet.
		}
	}

	// output last run.
	symbolsOut[outIndex] = symbol;
	countsOut[outIndex] = count;
	outIndex++;

	return outIndex;
}

void serial_rle(array<in_elt_t> in, std::vector<in_elt_t> &out_symbols, std::vector<int> &out_counts, int &out_end) {
	out_end = serial_rle_helper(in.data, in.size, out_symbols.data(), out_counts.data());
}

void inclusive_prefix_sum(array<uint8_t> d_in, array<int> d_out) {
    hipcub::CachingDeviceAllocator allocator(true);

    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    // Estimate temp_storage_bytes
    checkCuda(hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_in.data, d_out.data, d_in.size));
    checkCuda(allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));
    checkCuda(hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_in.data, d_out.data, d_in.size));
}

void paralel_rle_helper(array<in_elt_t> d_in, array<in_elt_t> d_out_symbols, array<int> d_out_counts, array<int> d_end) {
	auto d_backward_mask = array<uint8_t>::new_on_device(d_in.size);
	auto d_scanned_backward_mask = array<int>::new_on_device(d_in.size);
	auto d_compacted_backward_mask = array<int>::new_on_device(d_in.size + 1);

	hemi::parallel_for(0, d_backward_mask.size, [=] HEMI_LAMBDA(size_t i) {
		if (i == 0) {
			d_backward_mask.data[i] = 1;
			return;
		}
		d_backward_mask.data[i] = d_in.data[i] != d_in.data[i - 1];
	});
	inclusive_prefix_sum(d_backward_mask, d_scanned_backward_mask);	
	hemi::parallel_for(0, d_in.size, [=] HEMI_LAMBDA(size_t i) {
		if (i == 0) {
			d_compacted_backward_mask.data[i] = 0;
			return;
		}
		size_t out_pos = d_scanned_backward_mask.data[i] - 1;
		if (i == d_in.size - 1) {
			*d_end.data = out_pos + 1;
			d_compacted_backward_mask.data[out_pos + 1] = i + 1;
		}
		if (d_backward_mask.data[i])
			d_compacted_backward_mask.data[out_pos] = i;
	});
	
	// Not hemi::parallel_for because d_end is only on the device now.
	hemi::launch([=] HEMI_LAMBDA() {
		for (size_t i: hemi::grid_stride_range(0, *d_end.data)) {
			int current = d_compacted_backward_mask.data[i];
			int right = d_compacted_backward_mask.data[i + 1];
			d_out_counts.data[i] = right - current;
			d_out_symbols.data[i] = d_in.data[current];
		}
	});
	hemi::deviceSynchronize();

	d_compacted_backward_mask.hipFree();
	d_scanned_backward_mask.hipFree();
	d_backward_mask.hipFree();
}

void parallel_rle(array<in_elt_t> in, std::vector<in_elt_t> &out_symbols, std::vector<int> &out_counts, int &out_end) {
	auto d_in = array<in_elt_t>::new_on_device(in.size);
	auto d_out_symbols = array<in_elt_t>::new_on_device(in.size);
	auto d_out_counts = array<int>::new_on_device(in.size);
	auto d_end = array<int>::new_on_device(1);

	checkCuda(hipMemcpy(d_in.data, in.data, d_in.size * sizeof(*d_in.data), hipMemcpyHostToDevice));
	
	paralel_rle_helper(d_in, d_out_symbols, d_out_counts, d_end);

	checkCuda(hipMemcpy(out_symbols.data(), d_out_symbols.data, out_symbols.size() * sizeof(*out_symbols.data()), hipMemcpyDeviceToHost));
	checkCuda(hipMemcpy(out_counts.data(), d_out_counts.data, out_counts.size() * sizeof(*out_counts.data()), hipMemcpyDeviceToHost));
	checkCuda(hipMemcpy(&out_end, d_end.data, sizeof(out_end), hipMemcpyDeviceToHost));

	d_in.hipFree();
	d_out_symbols.hipFree();
	d_out_counts.hipFree();
	d_end.hipFree();
}

void run_rle_impl(array<in_elt_t> in, std::vector<in_elt_t> &out_symbols, std::vector<int> &out_counts, int &out_end, bool use_cpu_impl) {
	if (use_cpu_impl)
		serial_rle(in, out_symbols, out_counts, out_end);
  else
    parallel_rle(in, out_symbols, out_counts, out_end);
}

void rle(std::vector<in_elt_t> &in_owner, std::vector<in_elt_t> &full_out_symbols, std::vector<int> &full_out_counts, size_t piece_size, bool use_cpu_impl, bool verbose) {
	array<in_elt_t> full_in = array<in_elt_t>::vector_view_on_host(in_owner);

	for (size_t start = 0; start < in_owner.size(); start += piece_size) {
		array<in_elt_t> in = full_in.subview(start, piece_size);
		
		if(verbose)
			std::cout << "Partial in start: " << start << ", size: " << in.size << std::endl;

		// TODO Could actually be allocated once
		std::vector<in_elt_t> out_symbols(in.size);
		std::vector<int> out_counts(in.size);
		int end{0};

		run_rle_impl(in, out_symbols, out_counts, end, use_cpu_impl);

		out_symbols.resize(end);
		out_counts.resize(end);

		append_partial_result(out_symbols, out_counts, full_out_symbols, full_out_counts);
	}
}

void parse_input_args(int argc, char* argv[], size_t *input_size) {
    if(argc > 1) {
        *input_size = atoi(argv[1]);
    }
}

int get_single_digit_rand() {
	int singleDigit = rand() % 10;
	return singleDigit;
}

std::vector<in_elt_t> generate_input(size_t size) {
    std::vector<in_elt_t> res{};
    int multiplier = get_single_digit_rand();
    int value = get_single_digit_rand();
    for(int i = 0 ; i < size ; i++) {
        if(multiplier == 0) {
            multiplier = get_single_digit_rand();
            value = get_single_digit_rand();
        }
        res.push_back(value);
        multiplier--;
    }
    return res;
}

int main(int argc, char *argv[]) {
    srand(time(0));
    size_t input_size = 10000; //default input size
    size_t input_piece_size = 4;
    bool verbose = false;

    parse_input_args(argc, argv, &input_size);

    std::cout<<"Generating Input..."<<std::endl;
    std::vector<in_elt_t> input = generate_input(input_size);

    std::cout<<"Initial Input: "<<std::endl;
    std::cout<<"[";
    for(int i = 0 ; i < input.size() ; i++) {
        std::cout<<input[i]<<" ";
    }
    std::cout<<"]";
    std::cout<<std::endl;

    std::cout<<"Using the CPU implementation (Serial RLE Version)"<<std::endl;

    std::vector<in_elt_t> out_symbols{};
	std::vector<int> out_counts{};

    rle(input, out_symbols, out_counts, input_piece_size, true, verbose);

    std::cout<<"Output Symbols: "<<std::endl;
    std::cout<<"[";
    for(int i = 0 ; i < out_symbols.size() ; i++) {
        std::cout<<out_symbols[i]<<" ";
    }
    std::cout<<"]";
    std::cout<<std::endl;
    std::cout<<"Count: "<<std::endl;
    std::cout<<"[";
    for(int i = 0 ; i < out_counts.size() ; i++) {
        std::cout<<out_counts[i]<<" ";
    }
    std::cout<<"]";

    std::cout<<std::endl;
    std::cout<<"====================================================================="<<std::endl;

	std::cout<<"Using the GPU implementation (Parallel RLE Version)"<<std::endl;
	out_symbols.clear();
	out_counts.clear();

	rle(input, out_symbols, out_counts, input_piece_size, false, verbose);

    std::cout<<"Output Symbols: "<<std::endl;
    std::cout<<"[";
    for(int i = 0 ; i < out_symbols.size() ; i++) {
        std::cout<<out_symbols[i]<<" ";
    }
    std::cout<<"]";
    std::cout<<std::endl;
    std::cout<<"Count: "<<std::endl;
    std::cout<<"[";
    for(int i = 0 ; i < out_counts.size() ; i++) {
        std::cout<<out_counts[i]<<" ";
    }
    std::cout<<"]";
    std::cout<<std::endl;
    return 0;
}